#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<stdlib.h>
#include<math.h>
#include <time.h>
#define THREADS_PER_BLOCK 1024


__global__ void st(int* a, int* max, int *min,int *sum,int* n)
{
    __shared__ int smax[THREADS_PER_BLOCK];
    __shared__ int smin[THREADS_PER_BLOCK];
    __shared__ int ssum[THREADS_PER_BLOCK];
    int N = *n;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i < N)
      //   printf("%d\n", a[i]);
    if (i >= N)
        return;
    smax[tid] = a[i];
    smin[tid] = a[i];
    ssum[tid] = a[i];
    __syncthreads();
    for (unsigned int s = 1; s < blockDim.x && i < N; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            if ((tid + s) < blockDim.x)//a[i] < a[i + s]
            {
                if(smax[tid] < smax[tid + s])
                    smax[tid] = smax[tid + s];//sdata[tid] = sdata[tid + s];
                if (smin[tid] > smin[tid + s])
                    smin[tid] = smin[tid + s];
                ssum[tid] += ssum[tid + s];
            }
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        // printf("%d\n", sdata[0]);
        max[blockIdx.x] = smax[0];//sdata[0];
        min[blockIdx.x] = smin[0];
        sum[blockIdx.x] = ssum[0];
    }
}
__global__ void variance(int* a, double* var, double* av, int* n)
{
    __shared__ double svar[THREADS_PER_BLOCK];
    int N = *n;
    double mean = *av;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i < N)
      //   printf("%d\n", a[i]);
    if (i >= N)
        return;
    svar[tid] = pow(a[i] - mean, 2);
    __syncthreads();
    for (unsigned int s = 1; s < blockDim.x && i < N; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            if ((tid + s) < blockDim.x)//a[i] < a[i + s]
            {
                svar[tid] += svar[tid + s];
            }
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        // printf("%d\n", sdata[0]);
        var[blockIdx.x] = svar[0];//sdata[0];
    }
}
double* stats(int* a, int N)
{
    int max = a[0], min = a[0];
    double tot = 0;
    double* res = (double*)malloc(sizeof(double) * 4);
    for (int i = 0; i < N; i++)
    {
        if (a[i] > max)
            max = a[i];
        if (a[i] < min)
            min = a[i];
        tot += a[i];
    }
    res[0] = min;
    res[1] = max;
    res[2] = tot / N;
    res[3] = 0;
    for (int i = 0; i < N; i++)
    {
        res[3] += pow((a[i] - res[2]), 2);
    }
    res[3] = sqrt(res[3] / N);
    return res;
}
int main(int argc, char* argv[])
{
    clock_t start, end;
    double cpu_time_used;
    int N = 1e8;
    int size = N * (int)(sizeof(int));
    int *a, *d_a,*max,*d_max,*d_n,*min,*d_min,*mean,*d_mean;
    double* var, * d_var, * d_av;
    a = (int*)malloc(size); 
    srand(0);
    for (int i = 0; i < N; i++)
    {
        a[i] = rand() + rand() + rand();
        //printf("%d\t", a[i]);
    }
    printf("\n");
    start = clock();
    double* res = stats(a, N);
    end = clock();
    printf("Min=%f\tMax=%f\tAverage=%f\tVariance=%f\n", res[0], res[1], res[2], res[3]);
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Time elapsed in CPU implementation=%fs\n", cpu_time_used);
    int blocks = (int)ceil((double)N / THREADS_PER_BLOCK);
     max = (int*)malloc(blocks * sizeof(int));
     min = (int*)malloc(blocks * sizeof(int));
     mean = (int*)malloc(blocks * sizeof(int));
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_n, sizeof(int));
    hipMalloc((void**)&d_max, sizeof(int) * blocks);
    hipMalloc((void**)&d_min, sizeof(int) * blocks);
    hipMalloc((void**)&d_mean, sizeof(int) * blocks);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_n, &N, sizeof(int), hipMemcpyHostToDevice);
    start = clock();
    st <<<blocks, THREADS_PER_BLOCK >> > (d_a, d_max,d_min,d_mean, d_n);
    end = clock();
    cpu_time_used = ((double)(end - start));
    hipMemcpy(max, d_max, sizeof(int) * blocks, hipMemcpyDeviceToHost);
    hipMemcpy(min, d_min, sizeof(int) * blocks, hipMemcpyDeviceToHost);
    hipMemcpy(mean, d_mean, sizeof(int) * blocks, hipMemcpyDeviceToHost);
    hipFree(d_max); hipFree(d_min); hipFree(d_mean); 
    start = clock();
    int mx = max[0];
    int mn = min[0];
    double av = 0;
    for (int i = 0; i < blocks; i++)
    {
        if (max[i] > mx)
            mx = max[i];
        if (min[i] < mn)
            mn = min[i];
        av += mean[i];
    }
    av /= N;
    end = clock();
    cpu_time_used += ((double)(end - start));
    var = (double*)malloc(blocks * sizeof(double));
    hipMalloc((void**)&d_var, sizeof(double) * blocks);
    hipMalloc((void**)&d_av, sizeof(double));
    hipMemcpy(d_av, &av, sizeof(double), hipMemcpyHostToDevice);
    start = clock();
    variance << <blocks, THREADS_PER_BLOCK >> > (d_a, d_var, d_av,d_n);
    end = clock();
    cpu_time_used += ((double)(end - start));
    hipMemcpy(var, d_var, sizeof(double) * blocks, hipMemcpyDeviceToHost);
    hipFree(d_n); hipFree(d_av); hipFree(d_a); hipFree(d_var);
    double va = 0;
    start = clock();
    for (int i = 0; i < blocks; i++)
    {
        va += var[i];
    }
    va /= N;
    va = sqrt(va);
    end = clock();
    cpu_time_used += ((double)(end - start));
    cpu_time_used/= CLOCKS_PER_SEC;
    printf("--------------------------------------\n");
    printf("Min = % d\tMax = % d\tAverage = % f\tVariance = % f\n", mn, mx, av, va);
    printf("Time elapsed in GPU implementation=%fs\n", cpu_time_used);
    return 0;
}
