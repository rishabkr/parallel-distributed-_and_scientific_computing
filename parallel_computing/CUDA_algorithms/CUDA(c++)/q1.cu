#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<stdlib.h>
#include <cstdio>
#include <cstring>
#define THREADS_PER_BLOCK 1024
__global__ void iterate(float *room, float *h,int *n,int *iter)
{
	int N = *n;
	int it = *iter;
	int i = blockIdx.x;
	int j = threadIdx.x;		//indices i,j of h
	if (j >= N)
		return;
	int ir=i+1;
	int jr=j+1;	//corresponding indices in room
	for (int k = 0; k < it; k++)
	{
		h[i * N + j] = 0.25 * (room[(ir - 1) * (N + 2) + jr] + room[(ir + 1) * (N + 2) + jr] + room[ir * (N + 2) + jr - 1] + room[ir * (N + 2) + jr + 1]);
		__syncthreads();
		room[ir * (N + 2) + jr] = h[i * N + j];
		__syncthreads();
	}
}
int main()
{
	int N = 1000,it=100;
	int *d_n,*d_it;
	float* h, * room;
	h = (float*)malloc(sizeof(float) * N * N);
	room = (float*)malloc(sizeof(float) * (N+2) * (N+2));
	const int fireplace_start = 0.3 * N;
	const int fireplace_len = 0.4 * N;

	memset(h, 0, sizeof h);
	memset(room, 0, sizeof room);

	for (int i = 0; i < N + 2; ++i)
	{
		if (i > fireplace_start&& i <= fireplace_start + fireplace_len)
			room[i] = 100;
		else
			room[i] = 20;

		room[(N+1)*(N+2)+i] = 20;
	}

	for (int i = 1; i < N + 1; ++i)
		room[i*(N+2)] = room[i*(N+2)+(N + 1)] = 20;
	float *d_h, *d_room;
	hipMalloc((void**)&d_h, sizeof(float) *N*N);
	hipMalloc((void**)&d_n, sizeof(int));
	hipMalloc((void**)&d_it, sizeof(int));
	hipMalloc((void**)&d_room, sizeof(float) * (N+2) * (N+2));
	hipMemcpy(d_room, room, sizeof(float) * (N + 2) * (N + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &N, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_it,&it, sizeof(int), hipMemcpyHostToDevice);
	iterate<<<N, THREADS_PER_BLOCK >> > (d_room,d_h,d_n,d_it);
	hipMemcpy(h, d_h, sizeof(float) * N * N, hipMemcpyDeviceToHost);
	hipFree(d_n); hipFree(d_it); hipFree(d_h); hipFree(d_room);
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
			printf("%.2f ", h[i*(N)+j]);
		printf("\n");
	}
}
