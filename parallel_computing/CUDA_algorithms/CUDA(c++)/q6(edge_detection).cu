#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include<stdlib.h>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>

using namespace std;
using namespace cv;


__global__ void transform1st(float *a,float *b, int rows, int cols)
{
	//threadId -> Column 			blockId -> Row
	int self = threadIdx.x + blockIdx.x * blockDim.x;
	int l_u = ((threadIdx.x-1 >=0)&&(blockIdx.x-1 >=0)) ? ((threadIdx.x-1) + (blockIdx.x-1)* blockDim.x):-1;
	int l = (threadIdx.x-1 >=0) ? ((threadIdx.x-1) + blockIdx.x * blockDim.x):-1;
	int l_d = ((threadIdx.x-1 >=0)&&(blockIdx.x+1<rows)) ? ((threadIdx.x-1) + (blockIdx.x+1)* blockDim.x):-1;
	int r_u = ((threadIdx.x+1 < cols)&&(blockIdx.x-1>=0)) ? ((threadIdx.x+1) + (blockIdx.x-1)* blockDim.x):-1;
	int r = (threadIdx.x+1 < cols) ? ((threadIdx.x+1) + blockIdx.x * blockDim.x):-1;
	int r_d = ((threadIdx.x+1 <cols)&&(blockIdx.x+1 <rows)) ? ((threadIdx.x+1) + (blockIdx.x+1)* blockDim.x):-1;
	

	float luv = (l_u>=0) ? a[l_u] : 0;
	float lv = (l>=0) ? a[l] : 0;
	float ldv = (l_d>= 0) ? a[l_d] : 0;
	float ruv = (r_u >= 0) ? a[r_u] : 0;
	float rv = (r >= 0) ? a[r] : 0;
	float rdv = (r_d >= 0) ? a[r_d] : 0;
	
	b[self] = (-1*luv)+(-2*lv)+(-1*ldv)+(1*ruv)+(2*rv)+(1*rdv);
}

__global__ void transform2nd(float *a,float *b, int rows, int cols)
{
	//threadId -> Column 			blockId -> Row
	int self = threadIdx.x + blockIdx.x * blockDim.x;
	int l_u = ((threadIdx.x-1 >=0)&&(blockIdx.x-1 >=0)) ? ((threadIdx.x-1) + (blockIdx.x-1)* blockDim.x):-1;
	int u = (blockIdx.x-1 >=0) ? ((threadIdx.x) + (blockIdx.x-1) * blockDim.x):-1;
	int r_u = ((threadIdx.x+1 < cols)&&(blockIdx.x-1>=0)) ? ((threadIdx.x+1) + (blockIdx.x-1)* blockDim.x):-1;
	int l_d = ((threadIdx.x-1 >=0)&&(blockIdx.x+1<rows)) ? ((threadIdx.x-1) + (blockIdx.x+1)* blockDim.x):-1;
	int d = (blockIdx.x+1 <rows) ? ((threadIdx.x) + (blockIdx.x+1) * blockDim.x):-1;
	int r_d = ((threadIdx.x+1 <cols)&&(blockIdx.x+1 <rows)) ? ((threadIdx.x+1) + (blockIdx.x+1)* blockDim.x):-1;

	float luv = (l_u>=0) ? a[l_u] : 0;
	float uv = (u>=0) ? a[u] : 0;
	float ruv = (r_u >= 0) ? a[r_u] : 0;
	float ldv = (l_d>=0) ? a[l_d] : 0;
	float dv = (d >=0 ) ? a[d] : 0;
	float rdv = (r_d >= 0) ? a[r_d] : 0;

	b[self] = (-1*luv)+(-2*uv)+(-1*ruv)+(1*ldv)+(2*dv)+(1*rdv);
}

int main()
{
	Mat src, dst;
	float *src_a,*dst_a;
	src = imread("obh.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	//namedWindow("start");
    	//imshow("start", src);
	dst = src.clone();
	printf("Rows = %d\n",src.rows);
	printf("Columns = %d\n",src.cols);
	

	src_a = (float *)malloc(sizeof(float)*src.rows*src.cols);
	dst_a = (float *)malloc(sizeof(float)*src.rows*src.cols);
	for(int y = 0; y < src.rows; y++)
            for(int x = 0; x < src.cols; x++)
	    		src_a[y*src.cols+x] = src.at<uchar>(y,x);
	/*Declaring GPU Arrays*/
	float *src_g,*dst_g,*final;
	int size = sizeof(float)*src.rows*src.cols;
	hipMalloc((void **)&src_g,size);
	hipMalloc((void **)&dst_g,size);
	

	hipMalloc((void **)&final,size);
	hipMemcpy(src_g, src_a, size, hipMemcpyHostToDevice);
	transform1st<<<src.rows,src.cols>>>(src_g,dst_g,src.rows,src.cols);
	

	hipMemcpy(dst_a, dst_g, size, hipMemcpyDeviceToHost);
	hipMemcpy(src_g, src_a, size, hipMemcpyHostToDevice);
	

	transform2nd<<<src.rows,src.cols>>>(src_g,dst_g,src.rows,src.cols);
	hipMemcpy(src_a, dst_g, size, hipMemcpyDeviceToHost);
	hipMemcpy(src_g, src_a, size, hipMemcpyHostToDevice);
	hipMemcpy(dst_g, dst_a, size, hipMemcpyHostToDevice);
	

	for(int y = 0; y < src.rows; y++)
            for(int x = 0; x < src.cols; x++)
				dst.at<uchar>(y,x) = (uchar)dst_a[y*src.cols+x];
	/*Declaring GPU Arrays*/
	imwrite("obhfinal.jpg",dst);
	return 0;
}

